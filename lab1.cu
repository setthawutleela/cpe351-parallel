#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>

int main(int argc, char **argv){
    float *a_h, *b_h;   // Host data
    float *a_d, *b_d;   // Device data
    int N = 14, nBytes, i;

    printf("Start allocating\n");
    nBytes = N * sizeof(float);

    printf("Allocating in Host\n");
    a_h = (float*) malloc(nBytes);
    b_h = (float*) malloc(nBytes);

    printf("Allocating in Device\n");
    hipMalloc((void**)&a_d, nBytes);
    hipMalloc((void**)&b_d, nBytes);

    printf("End allocating\n");

    for(i=0; i<N; i++)
        a_h[i] = 100.0 + i;

    printf("Start memcpy\n");
    hipMemcpy(a_d, a_h, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(b_d, a_d, nBytes, hipMemcpyDeviceToDevice);
    hipMemcpy(b_h, b_d, nBytes, hipMemcpyDeviceToHost);
    
    printf("End memcpy\n");

    for(i=0; i<N; i++)
        assert(a_h[i] == b_h[i]);
    
    free(a_h);
    free(b_h);
    hipFree(a_d);
    hipFree(b_d);
	return 0;
}